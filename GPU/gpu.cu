#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/device_vector.h>

#define NUM_THREADS 8

const char *spgemm_desc = "GPU SpGEMM";

// -----------------------------------GLOBAL VARS--------------------------------
int blks;
int num_entries;
double *d_result;
int *d_A_row_ptrs;
int *d_B_row_ptrs;
int *d_A_col_indices;
int *d_B_col_indices;
double *d_A_values;
double *d_B_values;

// ----------------------------------DEVICE FUNCTIONS--------------------------

__global__ void spgemm_kernel(int *d_A_row_ptrs, int *d_A_col_indices, double *d_A_values,
                              int *d_B_row_ptrs, int *d_B_col_indices, double *d_B_values,
                              double *d_result, int rows_A, int cols_B)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= rows_A * cols_B)
        return;

    // We need to calculate the row and col from the thread id

    int row = tid / rows_A;
    int col = tid % cols_B;

    printf("This is the thread id %d, its row is %d, its col is %d \n", tid, row, col);

    double dot_prod = 0.0;

    printf("BEFORE THE FOR LOOP for thread %d, the row pointer is %d\n", tid, d_A_row_ptrs[row]);

    for (int k = d_A_row_ptrs[row]; k < d_A_row_ptrs[row + 1]; ++k)
    {
        int col_idx = d_A_col_indices[k];
        double A_val = d_A_values[k];

        for (int l = d_B_row_ptrs[col_idx]; l < d_B_row_ptrs[col_idx + 1]; ++l)
        {
            if (d_B_col_indices[l] == col)
            {
                double B_val = d_B_values[l];
                dot_prod += A_val * B_val;
                printf("HELL) THE value should be here %f * %f = %f\n", A_val, B_val, A_val * B_val);
                break;
            }
        }
    }

    printf("For thread id %d, the dot product is %f\n", tid, dot_prod);

    // We will store the value directly into the result array
    d_result[tid] = dot_prod;
    // if (dot_prod != 0.0)
    // {
    //     int index = atomicAdd(&result_row_ptrs[row + 1], 1);
    //     result_values[index] = dot_prod;
    //     result_col_indices[index] = col;
    // }
}

// -----------------------------------HOST FUNCTIONS--------------------------

void init_spgemm(const sparse_mat_t &A, const sparse_mat_t &B)
{
    num_entries = A.rows * B.cols;
    // The final matrix is of size rows and cols
    blks = (num_entries + NUM_THREADS - 1) / NUM_THREADS;

    hipMalloc(&d_result, sizeof(double) * num_entries);

    hipMalloc(&d_A_row_ptrs, sizeof(int) * (A.rows + 1));
    hipMemcpy(d_A_row_ptrs, A.row_ptrs.data(), sizeof(int) * (A.rows + 1), hipMemcpyHostToDevice);
    hipMalloc(&d_B_row_ptrs, sizeof(int) * (B.rows + 1));
    hipMemcpy(d_B_row_ptrs, B.row_ptrs.data(), sizeof(int) * (B.rows + 1), hipMemcpyHostToDevice);

    hipMalloc(&d_A_col_indices, sizeof(int) * (A.values.size()));
    hipMemcpy(d_A_col_indices, A.col_indices.data(), sizeof(int) * (A.values.size()), hipMemcpyHostToDevice);
    hipMalloc(&d_B_col_indices, sizeof(int) * (B.values.size()));
    hipMemcpy(d_B_col_indices, B.col_indices.data(), sizeof(int) * (B.values.size()), hipMemcpyHostToDevice);

    hipMalloc(&d_A_values, sizeof(double) * (A.values.size()));
    hipMemcpy(d_A_values, A.values.data(), sizeof(double) * (A.values.size()), hipMemcpyHostToDevice);
    hipMalloc(&d_B_values, sizeof(double) * (B.values.size()));
    hipMemcpy(d_B_values, B.values.data(), sizeof(double) * (B.values.size()), hipMemcpyHostToDevice);

    printf("The number of blocks %d \n", blks);
}

/*
 * This routine performs a sparse matrix multiplication operation
 * C := A * B
 * where A, B, and C are lda-by-lda matrices stored in column-major format.
 * On exit, A and B maintain their input values.
 */
sparse_mat_t spgemm(const sparse_mat_t &A, const sparse_mat_t &B)
{
    init_spgemm(A, B);

    // Fill the final result with all 0s
    thrust::device_ptr<double> d_result_ptr(d_result);
    thrust::fill(d_result_ptr, d_result_ptr + num_entries, 0);

    printf("BEFORE THE KERNEL\n");

    double *cpu_A_vals = new double[A.values.size()];
    hipMemcpy(cpu_A_vals, d_A_values, sizeof(double) * A.values.size(), hipMemcpyDeviceToHost);
    printf("printing A VALUES \n");
    printDoubleArray(A.values.data(), A.values.size());
    printDoubleArray(cpu_A_vals, A.values.size());

    // Parallelize the matrix multiplication across all the threads
    spgemm_kernel<<<blks, NUM_THREADS>>>(d_A_row_ptrs, d_A_col_indices, d_A_values,
                                         d_B_row_ptrs, d_B_col_indices, d_B_values,
                                         d_result, A.rows, B.cols);

    printf("AFTER THE KERNEL\n");

    // Copy the device result to host
    // Return a sparse mat representation from there

    double *result_cpu = new double[num_entries];

    hipMemcpy(result_cpu, d_result, sizeof(double) * num_entries, hipMemcpyDeviceToHost);

    printDoubleArray(result_cpu, num_entries);

    sparse_mat_t sparse_result = convert_to_sparse(A.rows, B.cols, result_cpu);
    return sparse_result;
}
